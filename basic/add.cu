#include "hip/hip_runtime.h"
#include<iostream>

using namespace std;

__global__ void add(int *a, int *b, int *c){
    *c = *a + *b;
}

int main(void){
    // input
    int a, b, c;
    // input to device
    int *d_a, *d_b, *d_c;
    int size = sizeof(int)

    // malloc d_a
    hipMalloc((void**) &d_a, size);
    hipMalloc((void**) &d_b, size);
    hipMalloc((void**) &d_c, size);

    a = 2;
    b = 7;

    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    add<<<1, 1>>>(d_a, d_b, d_c);
    cout << c << endl;

    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
