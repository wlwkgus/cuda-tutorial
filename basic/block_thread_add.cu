#include "hip/hip_runtime.h"
#include<iostream>

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

using namespace std;

__global__ void add(int *a, int *b, int *c, int n){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        c[index] = a[index] + b[index];
}

void random_ints(int* a, int n){
    for(int i=0; i < n; i++){
        a[i] = rand();
    }
}

int main(void){
    // input
    int *a, *b, *c;
    // input to device
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int)

    // malloc d_a
    hipMalloc((void**) d_a, size);
    hipMalloc((void**) d_b, size);
    hipMalloc((void**) d_c, size);

    a = (int *) malloc(size); random_ints(a, N)
    b = (int *) malloc(size); random_ints(a, N)
    c = (int *) malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<(N + THREADS_PER_BLOCK - 1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}
