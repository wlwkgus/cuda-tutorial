#include "hip/hip_runtime.h"
include <iostream>
#define BLOCK_SIZE = 512
#define RADIUS = 3

using namespace std;

__global__ void stencil_1d(int *in, int *out){
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    int gindex = threadIdx.x + blockIdx.x * blockDim.x;
    int lindex = threadIdx.x + RADIUS;

    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS) {
        temp[lindex - RADIUS] = in[gindex - RADIUS];
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }

    // To prevent WAW, RAW, WAR hazards, sync threads.
    __syncthreads();

    int result = 0;
    for(int offset=-RADIUS; offset <= RADIUS; offset++){
        result += temp[lindex + offset];
    }

    out[gindex] = result;
}
