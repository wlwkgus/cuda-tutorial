#include "hip/hip_runtime.h"
#include<iostream>

#define N 512
using namespace std;

__global__ void add(int *a, int *b, int *c){
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void random_ints(int* a, int N){
    int i;
    for(i=0; i < N; i++){
        a[i] = rand();
    }
}

int main(void){
    // input
    int *a, *b, *c;
    // input to device
    int *d_a, *d_b, *d_c;
    int size = sizeof(int);

    // malloc d_a
    hipMalloc((void**) &d_a, size);
    hipMalloc((void**) &d_b, size);
    hipMalloc((void**) &d_c, size);

    a = (int *) malloc(size); random_ints(a, N);
    b = (int *) malloc(size); random_ints(b, N);
    c = (int *) malloc(size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // use N blocks.
    add<<<N, 1>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);

    return 0;
}
