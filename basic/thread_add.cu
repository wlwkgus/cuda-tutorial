#include "hip/hip_runtime.h"
#include<iostream>

# define N 512

using namespace std;

__global__ void add(int *a, int *b, int *c){
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

int main(void){
    // input
    int *a, *b, *c;
    // input to device
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int)

    // malloc d_a
    hipMalloc((void**) &d_a, size);
    hipMalloc((void**) &d_b, size);
    hipMalloc((void**) &d_c, size);
    
    a = (int *) malloc(size); random_ints(a, N);
    b = (int *) malloc(size); random_ints(b, N);
    c = (int *) malloc(size);


    // Use N threads
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<1, N>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a); free(b); free(c);

    return 0;
}
